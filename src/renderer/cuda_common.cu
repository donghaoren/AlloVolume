#include "cuda_common.h"
#include <stdio.h>

namespace allovolume {

    void* gpuAllocate(size_t size) {
        void* result = 0;
        hipError_t err = hipMalloc(&result, size);
        if(!result) {
            fprintf(stderr, "cudaAllocate: hipMalloc() of %llu (%.2f MB): %s\n",
                size, size / 1048576.0,
                hipGetErrorString(err));
            size_t memory_free, memory_total;
            hipMemGetInfo(&memory_free, &memory_total);
            fprintf(stderr, "  Free: %.2f MB, Total: %.2f MB\n", (float)memory_free / 1048576.0, (float)memory_total / 1048576.0);
            throw bad_alloc();
        }
        return result;
    }

    void gpuDeallocate(void* pointer) {
        hipFree(pointer);
    }

    void gpuUpload(void* dest, const void* src, size_t size) {
        hipError_t err = hipMemcpy(dest, src, size, hipMemcpyHostToDevice);
        if(err != hipSuccess) {
            fprintf(stderr, "cudaUpload: hipMemcpy(): %s\n", hipGetErrorString(err));
            throw runtime_error();
        }
    }

    void gpuDownload(void* dest, const void* src, size_t size) {
        hipError_t err = hipMemcpy(dest, src, size, hipMemcpyDeviceToHost);
        if(err != hipSuccess) {
            fprintf(stderr, "cudaUpload: hipMemcpy(): %s\n", hipGetErrorString(err));
            throw runtime_error();
        }
    }

}
