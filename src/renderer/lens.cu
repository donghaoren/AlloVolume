#include "hip/hip_runtime.h"
#include "renderer.h"
#include <float.h>
#include <stdio.h>
#include <math_functions.h>
#include <algorithm>

#include "cuda_common.h"

namespace allovolume {

__global__
void get_rays_kernel(Lens::Ray* rays, int pixel_count, int width, int height, float focal_distance, float eye_separation) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx >= pixel_count) return;
    int x = idx % width;
    int y = idx / width;
    float theta = -((float)(x + 0.5) / width - 0.5f) * PI * 2;
    float phi = -((float)(y + 0.5) / height - 0.5f) * PI;
    Vector origin = Vector(0, 0, 0);
    Vector lookat = Vector(cos(theta) * cos(phi), sin(theta) * cos(phi), sin(phi)) * focal_distance;
    Vector shift = Vector(-sin(theta) * cos(phi), cos(theta) * cos(phi), 0) * eye_separation;
    origin += shift;
    rays[idx].origin = origin;
    rays[idx].direction = (lookat - origin).normalize();
}

class LensImpl_StereoAware : public Lens {
public:
    LensImpl_StereoAware() {
        eye_separation = 0.0f;
        focal_distance = 1.0f;
    }

    virtual void setParameter(const char* name, const void* value) {
        if(strcmp(name, "eye_separation") == 0) eye_separation = *(float*)value;
        if(strcmp(name, "focal_distance") == 0) focal_distance = *(float*)value;
    }

    float eye_separation, focal_distance;
};

class LensImpl_Equirectangular : public LensImpl_StereoAware {
public:
    LensImpl_Equirectangular() {
    }
    virtual void getRays(int width, int height, Ray* rays) {
        for(int y = 0; y < height; y++) {
            for(int x = 0; x < width; x++) {
                int p = y * width + x;
                float theta = -((float)(x + 0.5) / width - 0.5f) * PI * 2;
                float phi = -((float)(y + 0.5) / height - 0.5f) * PI;
                Vector origin = Vector(0, 0, 0);
                Vector lookat = Vector(cos(theta) * cos(phi), sin(theta) * cos(phi), sin(phi)) * focal_distance;
                Vector shift = Vector(-sin(theta) * cos(phi), cos(theta) * cos(phi), 0) * eye_separation;
                origin += shift;
                rays[p].origin = origin;
                rays[p].direction = (lookat - origin).normalize();
            }
        }
    }
    virtual void getRaysGPU(int width, int height, Ray* rays) {
        int number_of_threads = 64;
        int pixel_count = width * height;
        int n_blocks = diviur(pixel_count, number_of_threads);
        get_rays_kernel<<<n_blocks, number_of_threads>>>(rays, pixel_count, width, height, focal_distance, eye_separation);
    }
};

__global__
void get_rays_kernel_perspective(Lens::Ray* rays, int pixel_count, int width, int height, float focal_distance, float eye_separation, float screen_width) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx >= pixel_count) return;
    int x = idx % width;
    int y = idx / width;
    float kx = -((float)(x + 0.5) / width - 0.5f);
    float ky = -((float)(y + 0.5) / height - 0.5f) * ((float)height / (float)width);
    Vector origin = Vector(0, 0, 0);
    Vector lookat = Vector(1.0f, kx * screen_width, ky * screen_width) * focal_distance;
    Vector shift = Vector(0, eye_separation, 0);
    origin += shift;
    rays[idx].origin = origin;
    rays[idx].direction = (lookat - origin).normalize();
}

class LensImpl_Perspective : public LensImpl_StereoAware {
public:
    LensImpl_Perspective(float fovx_ = PI / 4.0f) : fovx(fovx_) { }

    virtual void getRays(int width, int height, Ray* rays) {
        float screen_width = tan(fovx / 2.0f) * 2.0f;
        for(int y = 0; y < height; y++) {
            for(int x = 0; x < width; x++) {
                int p = y * width + x;
                float kx = -((float)(x + 0.5) / width - 0.5f);
                float ky = -((float)(y + 0.5) / height - 0.5f) * ((float)height / (float)width);
                Vector origin = Vector(0, 0, 0);
                Vector lookat = Vector(1.0f, kx * screen_width, ky * screen_width) * focal_distance;
                Vector shift = Vector(0, eye_separation, 0);
                origin += shift;
                rays[p].origin = origin;
                rays[p].direction = (lookat - origin).normalize();
            }
        }
    }
    virtual void getRaysGPU(int width, int height, Ray* rays) {
        int number_of_threads = 64;
        int pixel_count = width * height;
        int n_blocks = diviur(pixel_count, number_of_threads);
        float screen_width = tan(fovx / 2.0f) * 2.0f;
        get_rays_kernel_perspective<<<n_blocks, number_of_threads>>>(rays, pixel_count, width, height, focal_distance, eye_separation, screen_width);
    }

    float fovx;
};

Lens* Lens::CreateEquirectangular() {
    return new LensImpl_Equirectangular();
}

Lens* Lens::CreatePerspective(float fovx) {
    return new LensImpl_Perspective(fovx);
}

Lens* Lens::CreateOrthogonal(float spanx) {

}

}
