#include "hip/hip_runtime.h"
#include "allovolume/renderer.h"
#include <float.h>
#include <stdio.h>
#include <math_functions.h>
#include <algorithm>

#include "cuda_common.h"

#include "rkv.h"

#include <sys/time.h>

#include "../timeprofiler.h"

#define PREINT_MAX_P 100

using namespace std;

namespace allovolume {

__device__
inline float interp(float a, float b, float t) {
    return fmaf(t, b - a, a);
}

__device__ __host__
inline int clampi(int value, int min, int max) {
    if(value < min) return min;
    if(value > max) return max;
    return value;
}

__device__ __host__
inline float clampf(float value, float min, float max) {
    return fmaxf(min, fminf(max, value));
}

__device__
inline float clamp01f(float value) { return __saturatef(value); }

__device__
inline Color tf_interpolate(Color* tf, int tf_size, float t) {
    float pos = clamp01f(t) * (tf_size - 1.0f);
    int idx = floor(pos);
    idx = clampi(idx, 0, tf_size - 2);
    float diff = pos - idx;
    Color t0 = tf[idx];
    Color t1 = tf[idx + 1];
    return t0 * (1.0 - diff) + t1 * diff;
}

struct transfer_function_t {
    Color* data;
    int size;

    inline __device__ Color get(float t) {
        return tf_interpolate(data, size, t);
    }
};

struct kd_tree_node_t {
    int split_axis; // 0, 1, 2 for x, y, z; -1 for leaf node, in leaf node, left = block index.
    float split_value;
    Vector bbox_min, bbox_max;
    int left, right;
};

struct ray_marching_parameters_t {
    const Lens::Ray* rays;
    Color* pixels;
    Color* pixels_back;
    VolumeRenderer::ClipRange* clip_ranges;

    Color bg_color;

    const BlockDescription* blocks;
    const float* data;
    int width, height;
    int block_count;
    float blend_coefficient;
    float step_size_multiplier;

    VolumeRenderer::RaycastingMethod raycasting_method;
    Vector bbox_min, bbox_max;

    const kd_tree_node_t* kd_tree;
    int kd_tree_root;

    int tf_size;

    Pose pose;
};

__device__ __host__
inline int intersectBox(Vector origin, Vector direction, Vector boxmin, Vector boxmax, float *tnear, float *tfar) {
    float tmin = FLT_MIN, tmax = FLT_MAX;
    float eps = 1e-8;
    if(fabs(direction.x) > eps) {
        float tx1 = (boxmin.x - origin.x) / direction.x;
        float tx2 = (boxmax.x - origin.x) / direction.x;
        tmin = fmaxf(tmin, fminf(tx1, tx2));
        tmax = fminf(tmax, fmaxf(tx1, tx2));
    } else {
        if(origin.x > boxmax.x || origin.x < boxmin.x) return false;
    }
    if(fabs(direction.y) > eps) {
        float ty1 = (boxmin.y - origin.y) / direction.y;
        float ty2 = (boxmax.y - origin.y) / direction.y;
        tmin = fmaxf(tmin, fminf(ty1, ty2));
        tmax = fminf(tmax, fmaxf(ty1, ty2));
    } else {
        if(origin.y > boxmax.y || origin.y < boxmin.y) return false;
    }
    if(fabs(direction.z) > eps) {
        float tz1 = (boxmin.z - origin.z) / direction.z;
        float tz2 = (boxmax.z - origin.z) / direction.z;
        tmin = fmaxf(tmin, fminf(tz1, tz2));
        tmax = fminf(tmax, fmaxf(tz1, tz2));
    } else {
        if(origin.z > boxmax.z || origin.z < boxmin.z) return false;
    }
    *tnear = tmin;
    *tfar = tmax;
    return tmax > tmin;
}

texture<float, 3, hipReadModeElementType> volume_texture;
texture<float4, 1, hipReadModeElementType> tf_texture;
texture<float4, 2, hipReadModeElementType> tf_texture_preintergrated;

struct block_interpolate_t {
    const float* data;
    float sx, sy, sz, tx, ty, tz;
    int cxsize, cysize, czsize;
    int ystride, zstride;

    __device__
    inline block_interpolate_t(const BlockDescription& block, const float* data_) {
        data = data_;
        sx = (block.xsize - block.ghost_count * 2.0f) / (block.max.x - block.min.x);
        sy = (block.ysize - block.ghost_count * 2.0f) / (block.max.y - block.min.y);
        sz = (block.zsize - block.ghost_count * 2.0f) / (block.max.z - block.min.z);
        tx = (float)block.ghost_count - 0.5f - block.min.x * sx;
        ty = (float)block.ghost_count - 0.5f - block.min.y * sy;
        tz = (float)block.ghost_count - 0.5f - block.min.z * sz;
        cxsize = block.xsize - 2;
        cysize = block.ysize - 2;
        czsize = block.zsize - 2;
        ystride = block.xsize;
        zstride = block.xsize * block.ysize;
    }

    __device__
    inline float interpolate(Vector pos) const {
        float px = fmaf(pos.x, sx, tx);
        float py = fmaf(pos.y, sy, ty);
        float pz = fmaf(pos.z, sz, tz);

        int ix = clampi(floor(px), 0, cxsize);
        int iy = clampi(floor(py), 0, cysize);
        int iz = clampi(floor(pz), 0, czsize);

        float tx = px - ix;
        float ty = py - iy;
        float tz = pz - iz;

        int idx = ix + ystride * iy + zstride * iz;

        float t00 = interp(data[idx], data[idx + zstride], tz);
        float t01 = interp(data[idx + ystride], data[idx + ystride + zstride], tz);
        float t0 = interp(t00, t01, ty);

        float t10 = interp(data[idx + 1], data[idx + 1 + zstride], tz);
        float t11 = interp(data[idx + 1 + ystride], data[idx + 1 + ystride + zstride], tz);
        float t1 = interp(t10, t11, ty);

        return interp(t0, t1, tx);
    }
};

struct block_interpolate_texture_t {
    Vector scale, translate;

    __device__
    inline float interpolate(Vector pos) const {
        float x = fmaf(pos.x, scale.x, translate.x);
        float y = fmaf(pos.y, scale.y, translate.y);
        float z = fmaf(pos.z, scale.z, translate.z);
        return tex3D(volume_texture, x, y, z);
    }
};

struct ray_marching_kernel_blockinfo_t {
    float kin, kout;
    int index;
};


__global__
void preprocess_data_kernel(float* data, float* data_processed, size_t data_size, TransferFunction::Scale scale, float min, float max) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx >= data_size) return;
    float value = data[idx];

    if(scale == TransferFunction::kLogScale) {
        if(value > 0) value = log(value);
        else value = min;
    }

    value = (value - min) / (max - min);

    data_processed[idx] = value;
}

__device__
inline Color tf_tex_get(float pos) {
    float4 f4 = tex1D(tf_texture, pos);
    return Color(f4.x, f4.y, f4.z, f4.w);
}

__device__
inline Color tf_tex_get2d(float px, float py) {
    float4 f4 = tex2D(tf_texture_preintergrated, px, py);
    return Color(f4.x, f4.y, f4.z, f4.w);
}

struct traverse_stack_t {
    int node;
    float kmin, kmax;
    int stage;
};

__device__
inline int kd_tree_block_intersection(
    Vector pos, Vector direction,
    float kmin, float kmax,
    float g_kin, float g_kout,
    const kd_tree_node_t* kd_tree, int kd_tree_root,
    const BlockDescription* blocks, ray_marching_kernel_blockinfo_t* blockinfos,
    traverse_stack_t* stack
) {
    int stack_pointer = 0;
    stack[0].node = kd_tree_root;
    stack[0].kmin = kmin;
    stack[0].kmax = kmax;
    stack[0].stage = 0;
    int blockinfos_count = 0;
    while(stack_pointer >= 0) {
        traverse_stack_t& s = stack[stack_pointer];
        int axis = kd_tree[s.node].split_axis;
        if(axis < 0) {
            float kin, kout;
            if(intersectBox(pos, direction, blocks[kd_tree[s.node].left].min, blocks[kd_tree[s.node].left].max, &kin, &kout)) {
                if(kin < g_kin) kin = g_kin;
                if(kin < kout) {
                    blockinfos[blockinfos_count].kin = kin;
                    blockinfos[blockinfos_count].kout = kout;
                    blockinfos[blockinfos_count].index = kd_tree[s.node].left;
                    blockinfos_count += 1;
                }
            }
            stack_pointer -= 1;
        } else {
            float split_value = kd_tree[s.node].split_value;
            float pmina = pos[axis] + direction[axis] * s.kmin;
            float pmaxa = pos[axis] + direction[axis] * s.kmax;
            if(pmina <= split_value && pmaxa <= split_value) {
                stack[stack_pointer].node = kd_tree[s.node].left;
                stack[stack_pointer].kmin = s.kmin;
                stack[stack_pointer].kmax = s.kmax;
            } else if(pmina >= split_value && pmaxa >= split_value) {
                stack[stack_pointer].node = kd_tree[s.node].right;
                stack[stack_pointer].kmin = s.kmin;
                stack[stack_pointer].kmax = s.kmax;
            } else {
                float k_split = (split_value - pos[axis]) / direction[axis];
                if(pmina < split_value) {
                    stack_pointer += 1;
                    stack[stack_pointer].node = kd_tree[s.node].right;
                    stack[stack_pointer].kmin = k_split;
                    stack[stack_pointer].kmax = s.kmax;
                    s.node = kd_tree[s.node].left;
                    s.kmax = k_split;
                } else {
                    stack_pointer += 1;
                    stack[stack_pointer].node = kd_tree[s.node].left;
                    stack[stack_pointer].kmin = k_split;
                    stack[stack_pointer].kmax = s.kmax;
                    s.node = kd_tree[s.node].right;
                    s.kmax = k_split;
                }
            }
        }
    }
    return blockinfos_count;
}

__global__
void ray_marching_kernel_basic(ray_marching_parameters_t p) {
    // Pixel index.
    int px = blockIdx.x * blockDim.x + threadIdx.x;
    int py = blockIdx.y * blockDim.y + threadIdx.y;
    if(px >= p.width || py >= p.height) return;
    register int idx = py * p.width + px;

    // Ray information.
    Lens::Ray ray = p.rays[idx];
    register Vector pos = p.pose.rotation.rotate(ray.origin) + p.pose.position;
    register Vector d = p.pose.rotation.rotate(ray.direction);

    // Initial color (background color).
    register Color color = p.bg_color;

    // Global ray information.
    float g_kin, g_kout;
    intersectBox(pos, d, p.bbox_min, p.bbox_max, &g_kin, &g_kout);
    if(g_kout < 0) {
        p.pixels[idx] = color;
        return;
    }
    if(g_kin < 0) g_kin = 0;

    // Block intersection.
    ray_marching_kernel_blockinfo_t blockinfos[128];
    traverse_stack_t stack[64];
    int blockinfos_count = kd_tree_block_intersection(pos, d, g_kin, g_kout, g_kin, g_kout, p.kd_tree, p.kd_tree_root, p.blocks, blockinfos, stack);

    // (Old O(n) Block intersection)
    // ray_marching_kernel_blockinfo_t blockinfos[512];
    // int blockinfos_count = 0;

    // for(int block_cursor = 0; block_cursor < p.block_count; block_cursor++) {
    //     BlockDescription block = p.blocks[block_cursor];
    //     float kin, kout;
    //     if(intersectBox(pos, d, block.min, block.max, &kin, &kout)) {
    //         if(kin < g_kin) kin = g_kin;
    //         if(kin < kout) {
    //             blockinfos[blockinfos_count].kin = kin;
    //             blockinfos[blockinfos_count].kout = kout;
    //             blockinfos[blockinfos_count].index = block_cursor;
    //             blockinfos_count += 1;
    //         }
    //     }
    // }

    // // Bubble-sort blocks according to distance.
    // for(;;) {
    //     bool swapped = false;
    //     int n = blockinfos_count;
    //     for(int c = 0; c < n - 1; c++) {
    //         if(blockinfos[c].kin < blockinfos[c + 1].kin) {
    //             ray_marching_kernel_blockinfo_t tmp = blockinfos[c + 1];
    //             blockinfos[c + 1] = blockinfos[c];
    //             blockinfos[c] = tmp;
    //             swapped = true;
    //         }
    //     }
    //     n -= 1;
    //     if(!swapped) break;
    // }

    // Simple solution: fixed step size.
    float kmax = g_kout;
    float L = p.blend_coefficient;

    // Render blocks.
    for(int cursor = 0; cursor < blockinfos_count; cursor++) {
        BlockDescription block = p.blocks[blockinfos[cursor].index];
        float kin = blockinfos[cursor].kin;
        float kout = blockinfos[cursor].kout;
        if(kout > kmax) kout = kmax;
        if(kin < kout) {
            // Render this block.
            float distance = kout - kin;
            float voxel_size = (block.max.x - block.min.x) / block.xsize; // assume voxels are cubes.
            int steps = ceil(distance / voxel_size / p.step_size_multiplier);
            if(steps > block.xsize * 30) steps = block.xsize * 30;
            if(steps < 2) steps = 2;
            float step_size = distance / steps;

            // Interpolate context.
            block_interpolate_t block_access(block, p.data + block.offset);

            // Blending with basic alpha compositing.
            for(int i = steps - 1; i >= 0; i--) {
                Color cm = tf_tex_get(block_access.interpolate(pos + d * (kin + step_size * ((float)i + 0.5f))));
                float k = expf(cm.a * step_size / L);
                color = Color(
                    cm.r * (1.0f - k) + color.r * k,
                    cm.g * (1.0f - k) + color.g * k,
                    cm.b * (1.0f - k) + color.b * k,
                    (1.0f - k) + color.a * k
                );
            }
            kmax = kin;
        }
    }

    // Un-premultiply alpha channel.
    if(color.a != 0) {
        color.r /= color.a;
        color.g /= color.a;
        color.b /= color.a;
    } else color = Color(0, 0, 0, 0);

    // Color output.
    p.pixels[idx] = color;
}

__global__
void ray_marching_kernel_preintegration(ray_marching_parameters_t p) {
    // Pixel index.
    int px = blockIdx.x * blockDim.x + threadIdx.x;
    int py = blockIdx.y * blockDim.y + threadIdx.y;
    if(px >= p.width || py >= p.height) return;
    register int idx = py * p.width + px;

    // Ray information.
    Lens::Ray ray = p.rays[idx];
    register Vector pos = p.pose.rotation.rotate(ray.origin) + p.pose.position;
    register Vector d = p.pose.rotation.rotate(ray.direction);
    register float k_front = FLT_MAX;
    register float k_far = FLT_MAX;
    if(p.clip_ranges) {
        float k_near = p.clip_ranges[idx].t_near;
        k_front = p.clip_ranges[idx].t_front;
        k_far = p.clip_ranges[idx].t_far;
        pos += d * k_near;
        k_front -= k_near;
        k_far -= k_near;
    }

    // Initial color (background color).
    register Color color = p.bg_color;

    // Global ray information.
    float g_kin, g_kout;
    intersectBox(pos, d, p.bbox_min, p.bbox_max, &g_kin, &g_kout);
    if(g_kout < 0) {
        if(p.pixels) p.pixels[idx] = color;
        if(p.pixels_back) p.pixels_back[idx] = color;
        return;
    }
    if(g_kin < 0) g_kin = 0;

    // Block intersection.
    ray_marching_kernel_blockinfo_t blockinfos[128];
    traverse_stack_t stack[64];
    int blockinfos_count = kd_tree_block_intersection(pos, d, g_kin, g_kout, g_kin, g_kout, p.kd_tree, p.kd_tree_root, p.blocks, blockinfos, stack);

    // Simple solution: fixed step size.
    float kmax = g_kout;
    float L = p.blend_coefficient;

    float tf_size = p.tf_size;

    // If rendering with front/back buffer, clamp to k_far, otherwise to k_front.
    bool is_rendering_back;
    if(p.pixels_back) {
        kmax = fminf(k_far, kmax);
        is_rendering_back = true;
    } else {
        kmax = fminf(k_front, kmax);
        is_rendering_back = false;
    }

    // Render blocks.
    for(int cursor = 0; cursor < blockinfos_count; cursor++) {
        BlockDescription block = p.blocks[blockinfos[cursor].index];
        float kin = blockinfos[cursor].kin;
        float kout = blockinfos[cursor].kout;
        if(kout > kmax) {
            kout = kmax;
        }
        bool is_back_finished = false;
        if(is_rendering_back) {
            if(kout <= k_front) {
                if(color.a != 0) {
                    color.r /= color.a;
                    color.g /= color.a;
                    color.b /= color.a;
                } else color = Color(0, 0, 0, 0);
                p.pixels_back[idx] = color;
                color = p.bg_color;
                is_rendering_back = false;
            } else if(kin <= k_front) {
                kin = k_front;
                is_back_finished = true;
            }
        }
        if(kin < kout) {
            // Render this block.
            float distance = kout - kin;
            float voxel_size = (block.max.x - block.min.x) / block.xsize; // assume voxels are cubes.
            int steps = ceil(distance / voxel_size / p.step_size_multiplier);
            if(steps > block.xsize * 30) steps = block.xsize * 30;
            if(steps <= 2) steps = 2;
            float step_size = distance / steps;

            // Blending with the pre-integrated lookup texture.
            // See "documents/allovolume-math.pdf" to see how we derived this.
            // Note that the formulas used in the code is a little bit more refined version,
            // They are essentially the same, although some terms are moved around for efficiency.
            // (We want to reduce the number of arithmetic operations in the rendering code).

            // The scaling factor of p.
            float pts_c = (step_size / L) / PREINT_MAX_P;
            // The minimum v0, v1 difference our pre-integration texture can tolerate.
            float mindiff = fmaxf(3.0 / tf_size, pts_c);

            // Interpolate context.
            block_interpolate_t block_access(block, p.data + block.offset);
            float val_prev = block_access.interpolate(pos + d * (kin + step_size * (float)steps));
            for(int i = steps - 1; i >= 0; i--) {
                // Access the volume.
                float val_this = block_access.interpolate(pos + d * (kin + step_size * (float)i));
                // Make sure val0 < val1 and val1 - val0 >= mindiff.
                float middle = (val_this + val_prev) / 2.0f;
                float diff = fmaxf(mindiff, fabs(val_this - val_prev)) / 2.0f;
                float val0 = middle - diff;
                float val1 = middle + diff;
                // Lookup the pre-integration table.
                float pts = pts_c / (val1 - val0);
                Color data0 = tf_tex_get2d(pts, val0);
                Color data1 = tf_tex_get2d(pts, val1);
                // Update the color.
                color.a = fmaf(data0.a, color.a, data1.a - data0.a) / data1.a;
                color.r = fmaf(data0.a, color.r, data1.r - data0.r) / data1.a;
                color.g = fmaf(data0.a, color.g, data1.g - data0.g) / data1.a;
                color.b = fmaf(data0.a, color.b, data1.b - data0.b) / data1.a;
                val_prev = val_this;
            }
            kmax = kin;
        }
        if(is_back_finished) {
            // Un-premultiply alpha channel.
            if(color.a != 0) {
                color.r /= color.a;
                color.g /= color.a;
                color.b /= color.a;
            } else color = Color(0, 0, 0, 0);
            p.pixels_back[idx] = color;
            // Reinitialize.
            is_rendering_back = false;
            color = p.bg_color;
            // Repeat the current block, since it's partially finished.
            cursor -= 1;
        }
    }

    // Un-premultiply alpha channel.
    if(color.a != 0) {
        color.r /= color.a;
        color.g /= color.a;
        color.b /= color.a;
    } else color = Color(0, 0, 0, 0);

    // Color output.
    if(is_rendering_back) {
        p.pixels_back[idx] = color;
        p.pixels[idx] = p.bg_color;
    } else {
        p.pixels[idx] = color;
    }
}

__global__
void ray_marching_kernel_rk4(ray_marching_parameters_t p) {
    // Pixel index.
    int px = blockIdx.x * blockDim.x + threadIdx.x;
    int py = blockIdx.y * blockDim.y + threadIdx.y;
    if(px >= p.width || py >= p.height) return;
    register int idx = py * p.width + px;

    // Ray information.
    Lens::Ray ray = p.rays[idx];
    register Vector pos = p.pose.rotation.rotate(ray.origin) + p.pose.position;
    register Vector d = p.pose.rotation.rotate(ray.direction);

    // Initial color (background color).
    register Color color = p.bg_color;

    // Global ray information.
    float g_kin, g_kout;
    intersectBox(pos, d, p.bbox_min, p.bbox_max, &g_kin, &g_kout);
    if(g_kout < 0) {
        p.pixels[idx] = color;
        return;
    }
    if(g_kin < 0) g_kin = 0;

    // Block intersection.
    ray_marching_kernel_blockinfo_t blockinfos[128];
    traverse_stack_t stack[64];
    int blockinfos_count = kd_tree_block_intersection(pos, d, g_kin, g_kout, g_kin, g_kout, p.kd_tree, p.kd_tree_root, p.blocks, blockinfos, stack);

    // Simple solution: fixed step size.
    float kmax = g_kout;
    float L = p.blend_coefficient;

    // Render blocks.
    for(int cursor = 0; cursor < blockinfos_count; cursor++) {
        BlockDescription block = p.blocks[blockinfos[cursor].index];
        float kin = blockinfos[cursor].kin;
        float kout = blockinfos[cursor].kout;
        if(kout > kmax) kout = kmax;
        if(kin < kout) {
            // Render this block.
            float distance = kout - kin;
            float voxel_size = (block.max.x - block.min.x) / block.xsize; // assume voxels are cubes.
            int steps = ceil(distance / voxel_size / p.step_size_multiplier);
            if(steps > block.xsize * 30) steps = block.xsize * 30;
            if(steps < 2) steps = 2;
            float step_size = distance / steps;

            // Interpolate context.
            block_interpolate_t block_access(block, p.data + block.offset);

            // Blending with RK4.
            Color c0 = tf_tex_get(block_access.interpolate(pos + d * kout));
            float c0s = c0.a / L;
            c0.a = 1.0f;
            for(int i = steps - 1; i >= 0; i--) {
                Color cm = tf_tex_get(block_access.interpolate(pos + d * (kin + step_size * ((float)i + 0.5f))));
                float cms = cm.a / L;
                cm.a = 1.0f;
                Color c1 = tf_tex_get(block_access.interpolate(pos + d * (kin + step_size * i)));
                float c1s = c1.a / L;
                c1.a = 1.0f;
                // Runge Kutta Order 4 method.
                // y'(t, y) = (y - c(t)) * ln(1 - alpha(t)) / L
                //   y has premultiplied alpha.
                //   c has non-premultiplied alpha.
                Color k1 = (color - c0) * c0s;
                Color k2 = (color + k1 * (step_size * 0.5f) - cm) * cms;
                Color k3 = (color + k2 * (step_size * 0.5f) - cm) * cms;
                Color k4 = (color + k3 * (step_size) - c1) * c1s;
                color = color + (k1 + (k2 + k3) * 2.0f + k4) * (step_size / 6.0f);

                c0 = c1;
                c0s = c1s;
            }
            kmax = kin;
        }
    }

    // Un-premultiply alpha channel.
    if(color.a != 0) {
        color.r /= color.a;
        color.g /= color.a;
        color.b /= color.a;
    } else color = Color(0, 0, 0, 0);

    // Color output.
    p.pixels[idx] = color;
}

struct render_dxdt_t {
    block_interpolate_t& block;
    Vector pos, d;
    double kin, kout;
    double L;
    __device__ render_dxdt_t(block_interpolate_t& block_, Vector pos_, Vector d_, double kin_, double kout_, double L_)
    : block(block_), pos(pos_), d(d_), kin(kin_), kout(kout_), L(L_) { }

    __device__ void operator() (double x, Color y, Color& dy) {
        // y'(t, y) = (y - c(t)) * ln(1 - alpha(t)) / L
        Color c = tf_tex_get(block.interpolate(pos + d * (kout - x)));
        double s = c.a / L;
        c.a = 1.0f;
        dy = (y - c) * s;
    }
};

struct color_norm_t {
    __device__ inline double operator() (Color c) {
        return fmax(fmax(fabs(c.r), fabs(c.g)), fmax(fabs(c.b), fabs(c.a)));
    }
};

__global__
void ray_marching_kernel_rkf_double(ray_marching_parameters_t p) {
    // Pixel index.
    int px = blockIdx.x * blockDim.x + threadIdx.x;
    int py = blockIdx.y * blockDim.y + threadIdx.y;
    if(px >= p.width || py >= p.height) return;
    register int idx = py * p.width + px;

    // Ray information.
    Lens::Ray ray = p.rays[idx];
    register Vector pos = p.pose.rotation.rotate(ray.origin) + p.pose.position;
    register Vector d = p.pose.rotation.rotate(ray.direction);

    // Initial color (background color).
    register Color color = p.bg_color;

    // Global ray information.
    float g_kin, g_kout;
    intersectBox(pos, d, p.bbox_min, p.bbox_max, &g_kin, &g_kout);
    if(g_kout < 0) {
        p.pixels[idx] = color;
        return;
    }
    if(g_kin < 0) g_kin = 0;

    // Block intersection.
    ray_marching_kernel_blockinfo_t blockinfos[128];
    traverse_stack_t stack[64];
    int blockinfos_count = kd_tree_block_intersection(pos, d, g_kin, g_kout, g_kin, g_kout, p.kd_tree, p.kd_tree_root, p.blocks, blockinfos, stack);

    // Simple solution: fixed step size.
    float kmax = g_kout;
    float L = p.blend_coefficient;

    // Render blocks.
    for(int cursor = 0; cursor < blockinfos_count; cursor++) {
        BlockDescription block = p.blocks[blockinfos[cursor].index];
        float kin = blockinfos[cursor].kin;
        float kout = blockinfos[cursor].kout;
        if(kout > kmax) kout = kmax;
        if(kin < kout) {
            // Render this block.
            float distance = kout - kin;
            float voxel_size = (block.max.x - block.min.x) / block.xsize; // assume voxels are cubes.
            block_interpolate_t block_access(block, p.data + block.offset);
            render_dxdt_t dxdt(block_access, pos, d, kin, kout, L);
            color_norm_t color_norm;
            Color new_color;
            RungeKuttaFehlberg(0.0f, distance, color, dxdt, color_norm, 1e-6f, voxel_size / 64.0f, voxel_size / 2.0f, new_color);
            color = new_color;
            kmax = kin;
        }
    }

    // Un-premultiply alpha channel.
    if(color.a != 0) {
        color.r /= color.a;
        color.g /= color.a;
        color.b /= color.a;
    } else color = Color(0, 0, 0, 0);

    // Color output.
    p.pixels[idx] = color;
}

__global__
void tf_preint_kernel(Color* table, Color* tf, float* Y, int size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i >= size) return;
    float rsum = 0, gsum = 0, bsum = 0;
    float p = (i + 0.5) / size * PREINT_MAX_P;
    for(int j = 0; j < size; j++) {
        int idx = j * size + i;
        float Y_j = Y[j];
        Color TF_j = tf[j];
        float scaler = -p * exp(-p * Y_j) * TF_j.a / size / 2;
        float dr = scaler * TF_j.r;
        float dg = scaler * TF_j.g;
        float db = scaler * TF_j.b;
        rsum += dr;
        gsum += dg;
        bsum += db;
        table[idx].r = rsum;
        table[idx].g = gsum;
        table[idx].b = bsum;
        table[idx].a = exp(-Y_j * p);
        rsum += dr;
        gsum += dg;
        bsum += db;
        __syncthreads();
    }
}

class VolumeRendererImpl : public VolumeRenderer {
public:

    VolumeRendererImpl() :
        blend_coefficient(1.0),
        step_size_multiplier(1.0),
        raycasting_method(kRK4Method),
        bbox_min(-1e20, -1e20, -1e20),
        bbox_max(1e20, 1e20, 1e20),
        bg_color(0, 0, 0, 0)
    {
        tf_texture_data = NULL;
        tf_texture_data_size = 0;
        tf_preint_preprocessed = false;
        floatChannelDesc = hipCreateChannelDesc<float>();
        image = NULL;
        image_back = NULL;
        clip_ranges_cpu = NULL;
    }

    struct BlockCompare {
        BlockCompare(Vector center_) {
            center = center_;
        }

        bool operator () (const BlockDescription& a, const BlockDescription& b) {
            double d1 = ((a.min + a.max) / 2.0f - center).len2_double();
            double d2 = ((b.min + b.max) / 2.0f - center).len2_double();
            return d1 > d2;
        }

        Vector center;
    };

    virtual void setBlendingCoefficient(float value) {
        blend_coefficient = value;
    }

    virtual void setVolume(VolumeBlocks* volume) {
        // Copy volume data.
        block_count = volume->getBlockCount();
        data.allocate(volume->getDataSize());
        data_processed.allocate(volume->getDataSize());
        data.upload(volume->getData());
        blocks.allocate(block_count);
        for(int i = 0; i < block_count; i++) {
            blocks[i] = *volume->getBlockDescription(i);
        }
        blocks.upload();
        buildKDTree();
    }

    void preprocessVolume() {
        float tf_min, tf_max;
        tf->getDomain(tf_min, tf_max);
        TransferFunction::Scale tf_scale = tf->getScale();
        // For non-linear scales, process the min, max values as well.
        if(tf_scale == TransferFunction::kLogScale) {
            tf_min = log(tf_min);
            tf_max = log(tf_max);
        }
        // Preprocess the volume.
        preprocess_data_kernel<<<diviur(data.size, 64), 64>>>(data.gpu, data_processed.gpu, data.size, tf->getScale(), tf_min, tf_max);
    }

    bool tf_preint_preprocessed;

    virtual void setTransferFunction(TransferFunction* tf_) {
        tf = tf_;
        tf_preint_preprocessed = false;
        if(raycasting_method == kPreIntegrationMethod) {
            uploadTransferFunctionPreintegratedGPU();
            tf_preint_preprocessed = true;
        }
    }

    virtual void setLens(Lens* lens_) {
        lens = lens_;
    }

    virtual void setImage(Image* image_) {
        image = image_;
    }

    virtual void setBackImage(Image* image_) {
        image_back = image_;
    }

    virtual void setPose(const Pose& pose_) {
        pose = pose_;
    }

    virtual void setStepSizeMultiplier(float value) {
        step_size_multiplier = value;
    }
    virtual float getStepSizeMultiplier() {
        return step_size_multiplier;
    }

    virtual void setBoundingBox(Vector min, Vector max) {
        bbox_min = min;
        bbox_max = max;
    }
    virtual void setRaycastingMethod(RaycastingMethod method) {
        raycasting_method = method;
    }

    virtual float getBlendingCoefficient() {
        return blend_coefficient;
    }
    virtual Pose getPose() {
        return pose;
    }
    virtual void getBoundingBox(Vector& min, Vector& max) {
        min = bbox_min;
        max = bbox_max;
    }
    virtual RaycastingMethod getRaycastingMethod() {
        return raycasting_method;
    }
    virtual void setBackgroundColor(Color color) {
        bg_color = color;
    }
    virtual Color getBackgroundColor() {
        return bg_color;
    }

    virtual void setClipRanges(ClipRange* ranges, size_t size) {
        clip_ranges_cpu = ranges;
        if(clip_ranges_cpu) {
            clip_ranges.allocate(size);
            clip_ranges.upload(clip_ranges_cpu);
            hipDeviceSynchronize();
        }
    }

    virtual void render() {
        render(0, 0, image->getWidth(), image->getHeight());
    }

    virtual void render(int x0, int y0, int total_width, int total_height) {
        // Prepare image.
        int pixel_count = image->getWidth() * image->getHeight();
        rays.allocate(pixel_count);

        // Generate rays.
        Lens::Viewport vp;
        vp.width = total_width;
        vp.height = total_height;
        vp.vp_x = x0; vp.vp_y = y0;
        vp.vp_width = image->getWidth(); vp.vp_height = image->getHeight();
        lens->getRaysGPU(vp, rays.gpu);

        ClipRange* clip_ranges_gpu = NULL;
        if(clip_ranges_cpu) {
            clip_ranges_gpu = clip_ranges.gpu;
        }

        // Proprocess the scale of the transfer function.
        preprocessVolume();

        // Upload the transfer function.
        if(raycasting_method == kPreIntegrationMethod) {
            if(!tf_preint_preprocessed) {
                uploadTransferFunctionPreintegratedGPU();
                tf_preint_preprocessed = true;
            }
        } else {
            uploadTransferFunctionTexture();
        }

        // Render kernel parameters.
        ray_marching_parameters_t pms;

        pms.rays = rays.gpu;
        pms.pixels = image ? image->getPixelsGPU() : NULL;
        pms.pixels_back = image_back ? image_back->getPixelsGPU() : NULL;
        pms.clip_ranges = clip_ranges_gpu;
        pms.blocks = blocks.gpu;
        pms.kd_tree = kd_tree.gpu;
        pms.kd_tree_root = kd_tree_root;
        pms.data = data_processed.gpu;
        pms.width = image->getWidth();
        pms.height = image->getHeight();
        pms.block_count = block_count;

        pms.bbox_min = bbox_min;
        pms.bbox_max = bbox_max;
        pms.raycasting_method = raycasting_method;

        // Other parameters.
        pms.blend_coefficient = blend_coefficient;
        pms.step_size_multiplier = step_size_multiplier;
        pms.bg_color = bg_color;
        // Block range.
        pms.pose = pose;

        pms.tf_size = tf->getSize();

        int blockdim_x = 8; // 8x8 is the optimal block size.
        int blockdim_y = 8;

        if(raycasting_method == kBasicBlendingMethod) {
            bindTransferFunctionTexture();
            ray_marching_kernel_basic<<<dim3(diviur(image->getWidth(), blockdim_x), diviur(image->getHeight(), blockdim_y), 1), dim3(blockdim_x, blockdim_y, 1)>>>(pms);
            unbindTransferFunctionTexture();
        }
        if(raycasting_method == kPreIntegrationMethod) {
            bindTransferFunctionTexture2D();
            ray_marching_kernel_preintegration<<<dim3(diviur(image->getWidth(), blockdim_x), diviur(image->getHeight(), blockdim_y), 1), dim3(blockdim_x, blockdim_y, 1)>>>(pms);
            unbindTransferFunctionTexture2D();
        }
        if(raycasting_method == kRK4Method) {
            bindTransferFunctionTexture();
            ray_marching_kernel_rk4<<<dim3(diviur(image->getWidth(), blockdim_x), diviur(image->getHeight(), blockdim_y), 1), dim3(blockdim_x, blockdim_y, 1)>>>(pms);
            unbindTransferFunctionTexture();
        }
        if(raycasting_method == kAdaptiveRKFMethod) {
            bindTransferFunctionTexture();
            ray_marching_kernel_rkf_double<<<dim3(diviur(image->getWidth(), blockdim_x), diviur(image->getHeight(), blockdim_y), 1), dim3(blockdim_x, blockdim_y, 1)>>>(pms);
            unbindTransferFunctionTexture();
        }
        hipDeviceSynchronize();
    }

    // Memory regions:
    MirroredMemory<BlockDescription> blocks;
    MirroredMemory<float> data, data_processed;
    MirroredMemory<Lens::Ray> rays;
    MirroredMemory<ClipRange> clip_ranges;
    ClipRange* clip_ranges_cpu;

    int block_count;
    TransferFunction* tf;
    Lens* lens;
    Image* image;
    Image* image_back;

    // Rendering parameters:
    Color bg_color;
    float blend_coefficient;
    float step_size_multiplier;
    RaycastingMethod raycasting_method;
    // Global bounding box:
    Vector bbox_min, bbox_max;
    // Pose:
    Pose pose;

    hipChannelFormatDesc floatChannelDesc;

    void uploadTransferFunctionTexture() {
        hipChannelFormatDesc channel_desc = hipCreateChannelDesc<float4>();

        if(tf_texture_data_size != tf->getSize()) {
            if(tf_texture_data) {
                hipFreeArray(tf_texture_data);
            }
            hipMallocArray(&tf_texture_data, &channel_desc, tf->getSize());
            tf_texture_data_size = tf->getSize();
        }

        Color* tf_color_logalpha = new Color[tf->getSize()];
        Color* tf_color = tf->getContent();
        for(int i = 0; i < tf->getSize(); i++) {
            tf_color_logalpha[i] = tf_color[i];
            tf_color_logalpha[i].a = log(1.0f - tf_color_logalpha[i].a);
        }

        hipMemcpyToArray(tf_texture_data, 0, 0,
            tf_color_logalpha,
            sizeof(float4) * tf->getSize(),
            hipMemcpyHostToDevice);

        delete [] tf_color_logalpha;

        tf_texture.normalized = 1;
        tf_texture.filterMode = hipFilterModeLinear;
        tf_texture.addressMode[0] = hipAddressModeClamp;
        tf_texture.addressMode[1] = hipAddressModeClamp;
    }

    MirroredMemory<float> preint_yt;
    MirroredMemory<Color> preint_tf, preint_table;

    void uploadTransferFunctionPreintegratedGPU() {
        hipChannelFormatDesc channel_desc = hipCreateChannelDesc<float4>();
        int size = tf->getSize();
        int size2d = size * size;

        if(tf_texture_data_size != size2d) {
            if(tf_texture_data) {
                hipFreeArray(tf_texture_data);
            }
            tf_texture_data = NULL;
            hipError_t err = hipMallocArray(&tf_texture_data, &channel_desc, size, size);
            if(!tf_texture_data) {
                int size_bytes = size * size * sizeof(Color);
                fprintf(stderr, "cudaAllocate: hipMalloc() of %d (%.2f MB): %s\n",
                    size_bytes, size_bytes / 1048576.0,
                    hipGetErrorString(err));
                size_t memory_free, memory_total;
                hipMemGetInfo(&memory_free, &memory_total);
                fprintf(stderr, "  Free: %.2f MB, Total: %.2f MB\n", (float)memory_free / 1048576.0, (float)memory_total / 1048576.0);
                throw bad_alloc();
            }
            tf_texture_data_size = size2d;
        }

        Color* tf_color = tf->getContent();

        preint_yt.allocate(size);
        preint_tf.allocate(size);
        preint_table.allocate(size2d);

        float csum = 0;
        for(int i = 0; i < size; i++) {
            float v = log(1.0f - tf_color[i].a);
            preint_tf[i] = tf_color[i];
            preint_tf[i].a = v;
            v /= size;
            csum += v / 2.0;
            preint_yt[i] = csum;
            csum += v / 2.0;
        }

        preint_yt.upload();
        preint_tf.upload();

        tf_preint_kernel<<<diviur(size, 64), 64>>>(preint_table.gpu, preint_tf.gpu, preint_yt.gpu, size);

        hipMemcpy2DToArray(tf_texture_data, 0, 0,
            preint_table.gpu,
            sizeof(float4) * size,
            sizeof(float4) * size, size,
            hipMemcpyDeviceToDevice);

        hipDeviceSynchronize();

        tf_texture_preintergrated.normalized = 1;
        tf_texture_preintergrated.filterMode = hipFilterModeLinear;
        tf_texture_preintergrated.addressMode[0] = hipAddressModeClamp;
        tf_texture_preintergrated.addressMode[1] = hipAddressModeClamp;
    }

    void bindTransferFunctionTexture2D() {
        hipChannelFormatDesc channel_desc = hipCreateChannelDesc<float4>();
        hipBindTextureToArray(tf_texture_preintergrated, tf_texture_data, channel_desc);
    }
    void unbindTransferFunctionTexture2D() {
        hipUnbindTexture(tf_texture_preintergrated);
    }
    void bindTransferFunctionTexture() {
        hipChannelFormatDesc channel_desc = hipCreateChannelDesc<float4>();
        hipBindTextureToArray(tf_texture, tf_texture_data, channel_desc);
    }
    void unbindTransferFunctionTexture() {
        hipUnbindTexture(tf_texture);
    }
    hipArray* tf_texture_data;
    size_t tf_texture_data_size;
    int tf_texture_preintergrated_steps;

    MirroredMemory<kd_tree_node_t> kd_tree;
    int kd_tree_root;
    int kd_tree_size;

    int buildKDTreeRecursive(kd_tree_node_t* nodes, int& nodes_count, int* blockids, int block_count, int axis) {
        if(block_count == 1) {
            kd_tree_node_t node;
            node.left = blockids[0];
            node.right = -1;
            node.split_value = 0;
            node.split_axis = -1;
            nodes[nodes_count++] = node;
            return nodes_count - 1;
        }
        float sp_min = FLT_MAX, sp_max = -FLT_MAX;
        for(int i = 0; i < block_count; i++) {
            sp_min = fminf(sp_min, blocks[blockids[i]].min[axis]);
            sp_max = fmaxf(sp_max, blocks[blockids[i]].max[axis]);
        }
        float split_value = (sp_min + sp_max) / 2.0f;

        int* blocks_left = new int[block_count];
        int blocks_left_count = 0;

        int* blocks_right = new int[block_count];
        int blocks_right_count = 0;

        for(int i = 0; i < block_count; i++) {
            if(blocks[blockids[i]].min[axis] + blocks[blockids[i]].max[axis] < 2.0f * split_value) {
                blocks_left[blocks_left_count++] = blockids[i];
            } else {
                blocks_right[blocks_right_count++] = blockids[i];
            }
        }

        // if(blocks_left_count == 0 || blocks_right_count == 0) {
        //     printf("Something wrong here.\n");
        //     printf("%f %f %f\n", sp_min, sp_max, split_value);
        //     for(int i = 0; i < block_count; i++) {
        //         printf("%f %f\n", blocks[blockids[i]].min[axis], blocks[blockids[i]].max[axis]);
        //     }
        //     exit(-1);
        // }

        int next_axis = (axis + 1) % 3;
        int left = buildKDTreeRecursive(nodes, nodes_count, blocks_left, blocks_left_count, next_axis);
        int right = buildKDTreeRecursive(nodes, nodes_count, blocks_right, blocks_right_count, next_axis);
        delete [] blocks_left;
        delete [] blocks_right;

        kd_tree_node_t node;
        node.left = left;
        node.right = right;
        node.split_value = split_value;
        node.split_axis = axis;
        nodes[nodes_count++] = node;
        return nodes_count - 1;
    }
    void buildKDTree() {
        kd_tree.allocate(blocks.size * 5);
        kd_tree_size = 0;

        int* blockids = new int[blocks.size];

        for(int i = 0; i < blocks.size; i++) blockids[i] = i;
        kd_tree_root = buildKDTreeRecursive(kd_tree.cpu, kd_tree_size, blockids, blocks.size, 0);
        kd_tree.size = kd_tree_size;
        kd_tree.upload();

        delete [] blockids;
    }
};

VolumeRenderer* VolumeRenderer::CreateGPU() {
    return new VolumeRendererImpl();
}

}
