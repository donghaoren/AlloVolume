#include "hip/hip_runtime.h"
#include "renderer.h"
#include <float.h>
#include <stdio.h>
#include <math_functions.h>
#include <algorithm>

#include "cuda_common.h"

#include "rkv.h"

// #define RKV_SOLVER

using namespace std;

namespace allovolume {

__device__
inline float interp(float a, float b, float t) {
    return fmaf(t, b - a, a);
}

__device__ __host__
inline int clampi(int value, int min, int max) {
    if(value < min) return min;
    if(value > max) return max;
    return value;
}

__device__ __host__
inline float clampf(float value, float min, float max) {
    return fmaxf(min, fminf(max, value));
}

__device__
inline float clamp01f(float value) { return __saturatef(value); }

__device__
inline Color tf_interpolate(Color* tf, float tf_min, float tf_max, int tf_size, float t) {
    float pos = clamp01f((t - tf_min) / (tf_max - tf_min)) * tf_size - 0.5f;
    int idx = floor(pos);
    idx = clampi(idx, 0, tf_size - 2);
    float diff = pos - idx;
    Color t0 = tf[idx];
    Color t1 = tf[idx + 1];
    return t0 * (1.0 - diff) + t1 * diff;
}

struct transfer_function_t {
    Color* data;
    int size;
    float min, max;
    TransferFunction::Scale scale;

    inline __device__ Color get(float t) {
        if(scale == TransferFunction::kLogScale) {
            if(t > 0) t = log(t);
            else t = min;
        }
        return tf_interpolate(data, min, max, size, t);
    }
};

struct ray_marching_parameters_t {
    const Lens::Ray* rays;
    Color* pixels;
    transfer_function_t tf;

    const BlockDescription* blocks;
    const float* data;
    int width, height;
    int block_count;
    float blend_coefficient;

    VolumeRenderer::RaycastingMethod raycasting_method;
    Vector bbox_min, bbox_max;

    Pose pose;
};

__device__ __host__
inline int intersectBox(Vector origin, Vector direction, Vector boxmin, Vector boxmax, float *tnear, float *tfar) {
    float tmin = FLT_MIN, tmax = FLT_MAX;
    float eps = 1e-8;
    if(fabs(direction.x) > eps) {
        float tx1 = (boxmin.x - origin.x) / direction.x;
        float tx2 = (boxmax.x - origin.x) / direction.x;
        tmin = fmaxf(tmin, fminf(tx1, tx2));
        tmax = fminf(tmax, fmaxf(tx1, tx2));
    } else {
        if(origin.x > boxmax.x || origin.x < boxmin.x) return false;
    }
    if(fabs(direction.y) > eps) {
        float ty1 = (boxmin.y - origin.y) / direction.y;
        float ty2 = (boxmax.y - origin.y) / direction.y;
        tmin = fmaxf(tmin, fminf(ty1, ty2));
        tmax = fminf(tmax, fmaxf(ty1, ty2));
    } else {
        if(origin.y > boxmax.y || origin.y < boxmin.y) return false;
    }
    if(fabs(direction.z) > eps) {
        float tz1 = (boxmin.z - origin.z) / direction.z;
        float tz2 = (boxmax.z - origin.z) / direction.z;
        tmin = fmaxf(tmin, fminf(tz1, tz2));
        tmax = fminf(tmax, fmaxf(tz1, tz2));
    } else {
        if(origin.z > boxmax.z || origin.z < boxmin.z) return false;
    }
    *tnear = tmin;
    *tfar = tmax;
    return tmax > tmin;
}

__device__
inline float access_volume(const float* data, int xsize, int ysize, int zsize, int ix, int iy, int iz) {
    return data[iz * xsize * ysize + iy * xsize + ix];
}

struct block_interpolate_t {
    const float* data;
    float sx, sy, sz, tx, ty, tz;
    int cxsize, cysize, czsize;
    int ystride, zstride;

    __device__
    inline block_interpolate_t(const BlockDescription& block, const float* data_) {
        data = data_;
        sx = (block.xsize - block.ghost_count * 2.0f) / (block.max.x - block.min.x);
        sy = (block.ysize - block.ghost_count * 2.0f) / (block.max.y - block.min.y);
        sz = (block.zsize - block.ghost_count * 2.0f) / (block.max.z - block.min.z);
        tx = (float)block.ghost_count - 0.5f - block.min.x * sx;
        ty = (float)block.ghost_count - 0.5f - block.min.y * sy;
        tz = (float)block.ghost_count - 0.5f - block.min.z * sz;
        cxsize = block.xsize - 2;
        cysize = block.ysize - 2;
        czsize = block.zsize - 2;
        ystride = block.xsize;
        zstride = block.xsize * block.ysize;
    }

    __device__
    inline float interpolate(Vector pos) const {
        float px = fmaf(pos.x, sx, tx);
        float py = fmaf(pos.y, sy, ty);
        float pz = fmaf(pos.z, sz, tz);

        int ix = clampi(floor(px), 0, cxsize);
        int iy = clampi(floor(py), 0, cysize);
        int iz = clampi(floor(pz), 0, czsize);

        float tx = px - ix;
        float ty = py - iy;
        float tz = pz - iz;

        int idx = ix + ystride * iy + zstride * iz;

        float t00 = interp(data[idx], data[idx + zstride], tz);
        float t01 = interp(data[idx + ystride], data[idx + ystride + zstride], tz);
        float t0 = interp(t00, t01, ty);

        float t10 = interp(data[idx + 1], data[idx + 1 + zstride], tz);
        float t11 = interp(data[idx + 1 + ystride], data[idx + 1 + ystride + zstride], tz);
        float t1 = interp(t10, t11, ty);

        return interp(t0, t1, tx);
    }
};

struct ray_marching_kernel_blockinfo_t {
    float kin, kout;
    int index;
};

struct render_dxdt_t {
    block_interpolate_t& block;
    transfer_function_t& tf;
    Vector pos, d;
    float kin, kout;
    float L;
    __device__ render_dxdt_t(block_interpolate_t& block_, transfer_function_t& tf_, Vector pos_, Vector d_, float kin_, float kout_, float L_)
    : block(block_), tf(tf_), pos(pos_), d(d_), kin(kin_), kout(kout_), L(L_) { }

    __device__ void operator() (float x, Color y, Color& dy) {
        // y'(t, y) = (y - c(t)) * ln(1 - alpha(t)) / L
        Color c = tf.get(block.interpolate(pos + d * (kout - x)));
        float s = log(1.0f - c.a) / L;
        c.a = 1.0f;
        dy = (y - c) * s;
    }
};

struct color_norm_t {
    __device__ inline float operator() (Color c) {
        return fmax(fmax(fabs(c.r), fabs(c.g)), fmax(fabs(c.b), fabs(c.a)));
    }
};

__global__
void ray_marching_kernel(ray_marching_parameters_t p) {
    // Pixel index.
    int px = blockIdx.x * blockDim.x + threadIdx.x;
    int py = blockIdx.y * blockDim.y + threadIdx.y;
    if(px >= p.width || py >= p.height) return;
    register int idx = py * p.width + px;

    // Ray information.
    Lens::Ray ray = p.rays[idx];
    register Vector pos = p.pose.rotation.rotate(ray.origin) + p.pose.position;
    register Vector d = p.pose.rotation.rotate(ray.direction);

    // Initial color (background color).
    register Color color(0, 0, 0, 0);

    // Global ray information.
    float g_kin, g_kout;
    intersectBox(pos, d, p.bbox_min, p.bbox_max, &g_kin, &g_kout);
    if(g_kout < 0) {
        p.pixels[idx] = color;
        return;
    }
    if(g_kin < 0) g_kin = 0;

    // Block intersection.
    ray_marching_kernel_blockinfo_t blockinfos[128];
    int blockinfos_count = 0;

    for(int block_cursor = 0; block_cursor < p.block_count; block_cursor++) {
        BlockDescription block = p.blocks[block_cursor];
        float kin, kout;
        if(intersectBox(pos, d, block.min, block.max, &kin, &kout)) {
            if(kin < g_kin) kin = g_kin;
            if(kin < kout) {
                blockinfos[blockinfos_count].kin = kin;
                blockinfos[blockinfos_count].kout = kout;
                blockinfos[blockinfos_count].index = block_cursor;
                blockinfos_count += 1;
            }
        }
    }

    // Bubble-sort blocks according to distance.
    for(;;) {
        bool swapped = false;
        int n = blockinfos_count;
        for(int c = 0; c < n - 1; c++) {
            if(blockinfos[c].kin < blockinfos[c + 1].kin) {
                ray_marching_kernel_blockinfo_t tmp = blockinfos[c + 1];
                blockinfos[c + 1] = blockinfos[c];
                blockinfos[c] = tmp;
                swapped = true;
            }
        }
        n -= 1;
        if(!swapped) break;
    }

    // Simple solution: fixed step size.
    float kmax = g_kout;
    float L = p.blend_coefficient;

    // Render blocks.
    for(int cursor = 0; cursor < blockinfos_count; cursor++) {
        BlockDescription block = p.blocks[blockinfos[cursor].index];
        float kin = blockinfos[cursor].kin;
        float kout = blockinfos[cursor].kout;
        if(kout > kmax) kout = kmax;
        if(kin < kout) {
            if(p.raycasting_method == VolumeRenderer::kAdaptiveRKVMethod) {
                // Render this block.
                float distance = kout - kin;
                float voxel_size = (block.max.x - block.min.x) / block.xsize; // assume voxels are cubes.
                block_interpolate_t block_access(block, p.data + block.offset);
                render_dxdt_t dxdt(block_access, p.tf, pos, d, kin, kout, L);
                color_norm_t color_norm;
                Color new_color;
                RungeKuttaVerner(0.0f, distance, color, dxdt, color_norm, 1e-6f, voxel_size / 10.0f, voxel_size, new_color);
                color = new_color;
            } else {
                // Render this block.
                float distance = kout - kin;
                float voxel_size = (block.max.x - block.min.x) / block.xsize; // assume voxels are cubes.
                int steps = ceil(distance / voxel_size);
                if(steps > block.xsize * 10) steps = block.xsize * 10;
                float step_size = distance / steps;

                // Interpolate context.
                block_interpolate_t block_access(block, p.data + block.offset);

                // Blending with RK4.
                Color c0 = p.tf.get(block_access.interpolate(pos + d * kout));
                float c0s = log(1.0f - c0.a) / L;
                c0.a = 1.0f;
                for(int i = steps - 1; i >= 0; i--) {
                    Color cm = p.tf.get(block_access.interpolate(pos + d * (kin + step_size * ((float)i + 0.5f))));
                    float cms = log(1.0f - cm.a) / L;
                    cm.a = 1.0f;
                    Color c1 = p.tf.get(block_access.interpolate(pos + d * (kin + step_size * i)));
                    float c1s = log(1.0f - c1.a) / L;
                    c1.a = 1.0f;
                    // Runge Kutta Order 4 method.
                    // y'(t, y) = (y - c(t)) * ln(1 - alpha(t)) / L
                    //   y has premultiplied alpha.
                    //   c has non-premultiplied alpha.
                    Color k1 = (color - c0) * c0s;
                    Color k2 = (color + k1 * (step_size * 0.5f) - cm) * cms;
                    Color k3 = (color + k2 * (step_size * 0.5f) - cm) * cms;
                    Color k4 = (color + k3 * (step_size) - c1) * c1s;
                    color = color + (k1 + (k2 + k3) * 2.0f + k4) * (step_size / 6.0f);

                    c0 = c1;
                    c0s = c1s;
                }
            }
            kmax = kin;
        }
    }

    // Un-premultiply alpha channel.
    if(color.a != 0) {
        color.r /= color.a;
        color.g /= color.a;
        color.b /= color.a;
    } else color = Color(0, 0, 0, 0);

    // Color output.
    p.pixels[idx] = color;
}

class VolumeRendererImpl : public VolumeRenderer {
public:

    VolumeRendererImpl() :
        blocks(512),
        volume_blocks(512),
        data(512 * 32 * 32 * 32),
        volume_data(512 * 34 * 34 * 34),
        rays(1000 * 1000),
        blend_coefficient(1.0),
        raycasting_method(kRK4Method),
        bbox_min(-1e20, -1e20, -1e20),
        bbox_max(1e20, 1e20, 1e20) { }

    struct BlockCompare {
        BlockCompare(Vector center_) {
            center = center_;
        }

        bool operator () (const BlockDescription& a, const BlockDescription& b) {
            double d1 = ((a.min + a.max) / 2.0f - center).len2_double();
            double d2 = ((b.min + b.max) / 2.0f - center).len2_double();
            return d1 > d2;
        }

        Vector center;
    };

    virtual void setBlendingCoefficient(float value) {
        blend_coefficient = value;
    }

    virtual void setVolume(VolumeBlocks* volume) {
        // Copy volume data.
        block_count = volume->getBlockCount();
        data.allocate(volume->getDataSize());
        data.upload(volume->getData());
        blocks.allocate(block_count);
        for(int i = 0; i < block_count; i++) {
            blocks[i] = *volume->getBlockDescription(i);
        }
    }

    virtual void setTransferFunction(TransferFunction* tf_) {
        tf = tf_;
    }

    virtual void setLens(Lens* lens_) {
        lens = lens_;
    }

    virtual void setImage(Image* image_) {
        image = image_;
    }

    virtual void setPose(const Pose& pose_) {
        pose = pose_;
    }

    virtual void setBoundingBox(Vector min, Vector max) {
        bbox_min = min;
        bbox_max = max;
    }
    virtual void setRaycastingMethod(RaycastingMethod method) {
        raycasting_method = method;
    }

    virtual void render() {
        // Sort blocks roughly.
        BlockCompare block_compare(pose.position);
        sort(blocks.cpu, blocks.cpu + block_count, block_compare);
        blocks.upload();

        // Prepare image.
        int pixel_count = image->getWidth() * image->getHeight();
        rays.allocate(pixel_count);

        // Generate rays.
        lens->getRaysGPU(image->getWidth(), image->getHeight(), rays.gpu);

        // Render kernel parameters.
        ray_marching_parameters_t pms;
        pms.rays = rays.gpu;
        pms.pixels = image->getPixelsGPU();
        pms.blocks = blocks.gpu;
        pms.data = data.gpu;
        pms.width = image->getWidth();
        pms.height = image->getHeight();
        pms.block_count = block_count;

        pms.bbox_min = bbox_min;
        pms.bbox_max = bbox_max;
        pms.raycasting_method = raycasting_method;

        // Set transfer function.
        pms.tf.data = tf->getContentGPU();
        tf->getDomain(pms.tf.min, pms.tf.max);
        pms.tf.scale = tf->getScale();
        pms.tf.size = tf->getSize();
        // For non-linear scales, process the min, max values as well.
        if(pms.tf.scale == TransferFunction::kLogScale) {
            pms.tf.min = log(pms.tf.min);
            pms.tf.max = log(pms.tf.max);
        }
        // Other parameters.
        pms.blend_coefficient = blend_coefficient;
        // Block range.
        pms.pose = pose;
        int blockdim_x = 8; // 8x8 is the optimal block size.
        int blockdim_y = 8;
        ray_marching_kernel<<<dim3(diviur(image->getWidth(), blockdim_x), diviur(image->getWidth(), blockdim_y), 1), dim3(blockdim_x, blockdim_y, 1)>>>(pms);
        hipDeviceSynchronize();
    }

    MirroredMemory<BlockDescription> blocks;
    MirroredMemory<float> volume_data, volume_blocks, data;
    MirroredMemory<Lens::Ray> rays;
    int block_count;
    TransferFunction* tf;
    Lens* lens;
    Image* image;
    Pose pose;
    float blend_coefficient;
    RaycastingMethod raycasting_method;
    Vector bbox_min, bbox_max;
};

VolumeRenderer* VolumeRenderer::CreateGPU() {
    return new VolumeRendererImpl();
}

}
